/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

// cjyeh
#define BLOCK_SIZE 1024

void check_param(void);
void update (void);
void printfinal (void);

int nsteps,                 	 /* number of time steps */
    tpoints, 	     		         /* total points along string */
    rcode;                  	 /* generic return code */

float  values[MAXPOINTS+2], 	 /* values at time t */
       old_val[MAXPOINTS+2], 	 /* values at time (t-dt) */
       new_val[MAXPOINTS+2]; 	 /* values at time (t+dt) */

// cjyeh
float  *values_d;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n",
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

/* run_parallel */
__global__ void run_parallel(float *values_d, int tpoints, int nsteps)
{
    int i, k;
    float x, fac, tmp;
    float dtime, c, dx, tau, sqtau;
    float value, new_val, old_val;

    /* init_line() */
    fac = 2.0 * PI;
    k = 1 + blockIdx.x * BLOCK_SIZE + threadIdx.x;
    tmp = tpoints - 1;
    x = (k - 1) / tmp;
    value = sin (fac * x);
    old_val = value;

    /* do_math() */
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    /* update() */
    if(k <= tpoints) {
      for (i = 1; i<= nsteps; i++) {
        if ((k == 1) || (k  == tpoints))
          new_val = 0.0;
        else
          new_val = (2.0 * value) - old_val + (sqtau * -2.0 * value);
        old_val = value;
        value = new_val;
      }
      values_d[k] = value;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i % 10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    // var
    int size;
    int block_num;

    sscanf(argv[1], "%d", &tpoints);
  	sscanf(argv[2], "%d", &nsteps);
  	check_param();

    size = (tpoints + 1) * sizeof(float);
    hipMalloc((void**) &values_d, size);

    printf("Initializing points on the line...\n");
  	//init_line();
  	printf("Updating all points for all time steps...\n");
  	//update();

    block_num = tpoints / BLOCK_SIZE + !(tpoints % BLOCK_SIZE == 0);

    run_parallel<<<block_num, BLOCK_SIZE>>>(values_d, tpoints, nsteps);

    hipMemcpy(values, values_d, size, hipMemcpyDeviceToHost);
    hipFree(values_d);

    printf("Printing final results...\n");
  	printfinal();
  	printf("\nDone.\n\n");

	return 0;
}
